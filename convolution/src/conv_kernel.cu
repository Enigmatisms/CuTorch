#include "hip/hip_runtime.h"
/**
 * CUDA convolution kernel
 * @author HQY @date 2021.9.26 
 */
#define __CUDA_INCLUDE_COMPILER_INTERNAL_HEADERS__
#include <hip/device_functions.h>
#include "../include/conv_kernel.h"

/**
 * @brief Convolution forward function
 * kernel is organized as grid(N, H, W) block(C_i, K, K)
 * @param output 
 * @return __global__ 
 */
__global__ void convForward(const float* const data, const float* const kernel, int out_chan, float* output) {
    const int gbase2 = gridDim.z, gbase1 = gridDim.y * gbase2, gbase0 = blockDim.x * gbase1;
    const int b = blockIdx.x, y = blockIdx.y, x = blockIdx.z, k_size = blockDim.y, half_k = (k_size >> 1);
    const int pos_y = threadIdx.y, pos_x = threadIdx.z, ch_id = threadIdx.x;
    const int ofst_x = pos_x - half_k + x, ofst_y = pos_y - half_k + y;
    const int k_sqr = k_size * k_size;
    const int id_offset = ch_id * k_sqr + pos_y * k_size + pos_x;
    const float val = data[gbase0 * b + gbase1 * ch_id + ofst_y * gbase2 + ofst_x];
    const int kbase = blockDim.x * k_sqr; 
    extern __shared__ float data_block[];       // k size * k size * sizeof(float) * in_channel + sizeof(float) * in_channel
    // kernel weight data is (C_o, C_i, K, K)
    for (int i = 0; i < out_chan; i++) {
        // kernel is too big to fit into the shared memory (occupancy & latency hiding)
        float tmp = kernel[i * kbase + id_offset] * val;        
        data_block[ch_id] += tmp;
        __syncthreads();                    // wait till c_i, k, k threads are all done
        if ((ch_id | x | y) == 0)           // execute only once (warp divergence)
            output[b * out_chan * gbase1 + gbase1 * i + gbase2 * y + x] = tmp;
        __syncthreads();
        // bias is added in another kernel function
    }
}

/// @note grid will be (N, H, W), block shall be 1-dim (C)
__global__ void biasForward(const float* const data, const float* const bias, float* output) {
    const int gbase2 = blockDim.x, gbase1 = gridDim.z * gbase2, gbase0 = gridDim.y * gbase1;
    const int id = threadIdx.x, base = blockIdx.x * gbase0 + id * gbase1 + blockIdx.y * gbase2 + blockIdx.z;
    output[base] = data[base] + bias[id];
}


/// @note 输入是(N, C_i, H, W)以及(C_o, C_i, K, K)的卷积核,输出是(N, C_O, H, W)
/// @note 注意，输入的data进行了padding，而输出output没有padding，不考虑stride
/// @ 输入图像从pad开始 到x+pad结束
__global__ void convForwardV2(
    ConstFloatPtr data, ConstFloatPtr kernel, ConstFloatPtr bias, 
    FloatPtr output, const int ks, const int co_num
) {
    /// 数据复制到shared memory下 注意，这里有两部分
    /// 一部分是原始数据，另一部分是累加结果
    /// 一个函数只处理batch中的一张图片中的一个点(x, y)
    extern __shared__ float proc[];
    const int n = blockIdx.x, y = blockIdx.y, x = blockIdx.z, c = threadIdx.x, k = threadIdx.y, hks = (ks / 2);
    const int row_offset = gridDim.z + (hks << 1), chan_offset = (gridDim.y + hks << 1) * row_offset;
    const int row_base = (y + hks) * row_offset;
    const int p_row_offset = ks, p_chn_offset = ks * ks, co_offset = blockDim.x * p_chn_offset;
    const int full_base = n * chan_offset * blockDim.x + c * chan_offset + row_base + x + hks, 
              p_full_base = c * p_chn_offset + k * p_row_offset;
    FloatPtr co_output = &proc[co_num * co_offset];     // 位置需要指定
    for (int i = -hks; i <= hks; i++) {
        proc[p_full_base + i] = data[full_base + (k - hks) * row_offset + i];
    }
    FloatPtr ptr = co_output;
    for (int i = 0; i < co_num; i++) {
        ConstFloatPtr ki = &kernel[i * co_offset];
        for (int j = 0; j < ks; j++) {
            float val = ki[p_full_base + j] * proc[p_full_base + j];
            atomicAdd_system(ptr, val);
        }
        ptr++;
    }
    __syncthreads();
    // 从共享内存复制到global内存
    const int row_base_no_pad = y * gridDim.z;
    if (k == 0) {       // warp divergence
        const int obatch_base = n * co_num * chan_offset;
        for (int i = 0; i < 4; i++) {
            const int id = blockDim.x * i + c;
            if (id >= co_num) break;
            output[obatch_base + id * chan_offset + row_base_no_pad + x] = co_output[id] + bias[id];
        }
    }
}

/// 所以，设计应该是：<<<(N, C_o,  K * K), (C_i, H, W)>>>, 输入的x经过padding
__global__ void convBackwardForW(
    ConstFloatPtr grad_upstream, ConstFloatPtr x, FloatPtr grad_w, const int k
) {
    extern __shared__ float all_ci[];
    const int hk = (k >> 1);
    const int r_offset = blockDim.z, r_offset_p = r_offset + (hk << 1),
        c_offset = blockDim.y * r_offset, c_offset_p = (blockDim.y + (hk << 1) * r_offset_p),
        b_offset = blockDim.x * c_offset, b_offset_p = blockDim.x * c_offset_p;
    const int k2 = k * k, wc_offset = blockDim.x * k2;
    const int ci = threadIdx.x, n = blockIdx.x, co = blockIdx.y, w_id = blockIdx.z, k_row = w_id / k, k_col = w_id % k;
    FloatPtr this_ci = &all_ci[ci];
    float val = x[n * b_offset_p + ci * c_offset_p + (threadIdx.y + k_row) * r_offset_p + k_col + threadIdx.z] *
        grad_upstream[n * b_offset + co * c_offset + threadIdx.y * r_offset + threadIdx.z];
    atomicAdd_system(this_ci, val);
    __syncthreads();
    if (threadIdx.y == 0 && threadIdx.z == 0) {
        grad_w[co * wc_offset + ci * k2 + w_id] = all_ci[ci];
    }
}

__global__ void convBackwardForX(
    
) {
    ;
}
