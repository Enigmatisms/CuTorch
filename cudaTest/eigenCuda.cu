#include "hip/hip_runtime.h"
#include <chrono>
#include <vector>
#include <iostream>
#include <Eigen/Dense>
#include <hip/hip_runtime.h>
#include <>

typedef Eigen::Matrix<double, 16, 12> MatrixXd;

__global__ void matMul(const double const* src1, const double const* src2, double* dst) {
    int y = blockIdx.x, x = threadIdx.x;
    int rows = gridDim.x, cols = blockDim.x;
    int id = y * cols + x;
    dst[id] = src1[id] * src2[id];
}

__global__ void eigenDirect(const MatrixXd* const src1, const MatrixXd* const src2, MatrixXd* dst) {
    int y = blockIdx.x, x = threadIdx.x;
    int rows = gridDim.x, cols = blockDim.x;
    dst->operator()(y, x) = src1->operator()(y, x) * src2->operator()(y, x);
}


__device__ double determinant(const Eigen::Matrix2d& mat) {
    return mat(0, 0) * mat(1, 1) - mat(1, 0) * mat(0, 1);
}

__device__ Eigen::Matrix2d matInverse(Eigen::Matrix2d mat) {
    double tmp1 = mat(0, 0), tmp2 = mat(1, 0);
    double det = determinant(mat);
    if (abs(det) > 1e-5) {
        mat(0, 0) = mat(1, 1);
        mat(1, 0) = -mat(0, 1);
        mat(0, 1) = -tmp2;
        mat(1, 1) = tmp1;
        return mat / det;
    }
    return Eigen::Matrix2d::Zero();
}

__global__ void streamEigenProcess(const Eigen::Matrix2d* const src1, const Eigen::Vector2d* const src2, Eigen::Vector2d* dst) {
    int id = threadIdx.x;
    Eigen::Matrix2d inv = matInverse(src1[id]);
    dst[id] = inv * src2[id];
}

int main() {
    // ======================= test 1 ========================
    int row = 16, col = 12;
    Eigen::Matrix<double, 16, 12> A;
    A.setRandom();
    Eigen::Matrix<double, 16, 12> B;
    B.setRandom();
    Eigen::Matrix<double, 16, 12> C;
    C.setZero();
    double *dev_a, *dev_b, *dev_c;
    size_t mat_size = sizeof(double) * row * col;
    hipMalloc((void **) &dev_a, mat_size);
    double start_t = std::chrono::system_clock::now().time_since_epoch().count() / 1e6;
    hipMalloc((void **) &dev_b, mat_size);
    hipMalloc((void **) &dev_c, mat_size);
    double end_t = std::chrono::system_clock::now().time_since_epoch().count() / 1e6;
    printf("malloc time: %lf ms\n", end_t - start_t);
    hipMemcpy(dev_a, A.data(), mat_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, B.data(), mat_size, hipMemcpyHostToDevice);
    matMul <<< row, col >>> (dev_a, dev_b, dev_c);
    hipMemcpy(C.data(), dev_c, mat_size, hipMemcpyDeviceToHost);
    // std::cout << A << std::endl << std::endl;
    // std::cout << B << std::endl << std::endl;
    // std::cout << C << std::endl << std::endl;
    start_t = std::chrono::system_clock::now().time_since_epoch().count() / 1e6;
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    end_t = std::chrono::system_clock::now().time_since_epoch().count() / 1e6;
    printf("free time: %lf ms\n", end_t - start_t);

    // ======================= test 2 ========================
    // Eigen::Matrix<double, 16, 12> *mat_a, *mat_b, *mat_c;
    // size_t eigen_size = sizeof(A);
    // hipMalloc((void **) &mat_a, eigen_size);
    // hipMalloc((void **) &mat_b, eigen_size);
    // hipMalloc((void **) &mat_c, eigen_size);
    // C.setZero();
    // hipMemcpy(mat_a->data(), A.data(), mat_size, hipMemcpyHostToDevice);
    // hipMemcpy(mat_b->data(), B.data(), mat_size, hipMemcpyHostToDevice);
    // eigenDirect <<< row, col >>> (mat_a, mat_b, mat_c);
    // hipMemcpy(C.data(), mat_c->data(), mat_size, hipMemcpyDeviceToHost);
    // std::cout << C << std::endl << std::endl;
    // hipFree(mat_a);
    // hipFree(mat_b);
    // hipFree(mat_c);

    // std::vector<Eigen::Matrix2d> mats;
    // std::vector<Eigen::Vector2d> vecs;
    // std::vector<Eigen::Vector2d> result(128, Eigen::Vector2d::Zero());
    // for (int i = 0; i < 128; i++) {
    //     mats.push_back(Eigen::Matrix2d::Random());
    //     vecs.push_back(Eigen::Vector2d::Random());
    // }
    // Eigen::Matrix2d* mat_ptr;
    // Eigen::Vector2d* vec_ptr, *res_ptr;
    // size_t mats_size = sizeof(Eigen::Matrix2d) * 128;
    // size_t vecs_size = sizeof(Eigen::Vector2d) * 128;
    // hipMalloc((void **) &mat_ptr, mats_size);
    // hipMalloc((void **) &vec_ptr, vecs_size);
    // hipMalloc((void **) &res_ptr, vecs_size);
    // hipMemcpy(mat_ptr, mats.data(), mats_size, hipMemcpyHostToDevice);
    // hipMemcpy(vec_ptr, vecs.data(), vecs_size, hipMemcpyHostToDevice);
    // streamEigenProcess <<< 1, 128 >>> (mat_ptr, vec_ptr, res_ptr);
    // hipMemcpy(result.data(), res_ptr, vecs_size, hipMemcpyDeviceToHost);
    // hipFree(mat_ptr);
    // hipFree(vec_ptr);
    // hipFree(res_ptr);
    // for (int i = 0; i < 128; i++) {
    //     std::cout << result[i] << std::endl << std::endl;
    // }
    return 0;
}